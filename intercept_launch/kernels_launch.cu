#include "interceptor.h"
#include <stdio.h>
#include <stdlib.h>

extern hipFunction_t matrixAdd_func;
extern hipFunction_t matrixSub_func;
extern hipFunction_t matrixMul_func;

// 包装函数实现
void launchMatrixAdd(dim3 gridDim, dim3 blockDim, hipStream_t stream,
                    hipDeviceptr_t d_A, hipDeviceptr_t d_B, hipDeviceptr_t d_C, int N) {
    currentKernelType = KERNEL_MATRIX_ADD;
    currentKernelFunc = matrixAdd_func;

    // 准备内核参数
    void *args[] = { &d_A, &d_B, &d_C, &N };

    // 启动内核
    hipError_t result = hipModuleLaunchKernel(
        currentKernelFunc,
        gridDim.x, gridDim.y, gridDim.z,
        blockDim.x, blockDim.y, blockDim.z,
        0, // sharedMem
        stream,
        args,
        nullptr // extra
    );

    if (result != hipSuccess) {
        const char *errorStr;
        hipDrvGetErrorString(result, &errorStr);
        //fprintf(stderr, "Failed to launch matrixAdd: %s\n", errorStr);
    };
}

void launchMatrixSub(dim3 gridDim, dim3 blockDim, hipStream_t stream,
                    hipDeviceptr_t d_A, hipDeviceptr_t d_B, hipDeviceptr_t d_C, int N) {
    currentKernelType = KERNEL_MATRIX_SUB;
    currentKernelFunc = matrixSub_func;

    // 准备内核参数
    void *args[] = { &d_A, &d_B, &d_C, &N };

    // 启动内核
    hipError_t result = hipModuleLaunchKernel(
        currentKernelFunc,
        gridDim.x, gridDim.y, gridDim.z,
        blockDim.x, blockDim.y, blockDim.z,
        0, // sharedMem
        stream,
        args,
        nullptr // extra
    );

    if (result != hipSuccess) {
        const char *errorStr;
        hipDrvGetErrorString(result, &errorStr);
        //fprintf(stderr, "Failed to launch matrixSub: %s\n", errorStr);
    };
}

void launchMatrixMul(dim3 gridDim, dim3 blockDim, hipStream_t stream,
                    hipDeviceptr_t d_A, hipDeviceptr_t d_B, hipDeviceptr_t d_C, int N) {
    currentKernelType = KERNEL_MATRIX_MUL;
    currentKernelFunc = matrixMul_func;

    // 准备内核参数
    void *args[] = { &d_A, &d_B, &d_C, &N };

    // 启动内核
    hipError_t result = hipModuleLaunchKernel(
        currentKernelFunc,
        gridDim.x, gridDim.y, gridDim.z,
        blockDim.x, blockDim.y, blockDim.z,
        0, // sharedMem
        stream,
        args,
        nullptr // extra
    );

    if (result != hipSuccess) {
        const char *errorStr;
        hipDrvGetErrorString(result, &errorStr);
       // fprintf(stderr, "Failed to launch matrixMul: %s\n", errorStr);
    };
}
