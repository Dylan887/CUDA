#include "interceptor.h"
#include <stdio.h>
#include <stdlib.h>


// 封装的加载函数
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



// 封装的加载模块及内核函数的函数
void loadModuleAndFunctions(const char *modulePath, KernelFunction *kernelFunctions,int numKernels) {
    // 加载模块
    hipModule_t module;
    hipError_t result = hipModuleLoad(&module, modulePath);
    if (result != hipSuccess) {
        const char *errorStr;
        hipDrvGetErrorString(result, &errorStr);
        fprintf(stderr, "Failed to load CUDA module: %s\n", errorStr);
        exit(EXIT_FAILURE);
    }
    printf("Successfully loaded module: %s\n", modulePath);
    
    // 加载内核函数指针
    for (int i = 0; i < numKernels; i++) {
        result = hipModuleGetFunction(kernelFunctions[i].funcPtr, module, kernelFunctions[i].funcName);
        if (result != hipSuccess) {
            const char *errorStr;
            hipDrvGetErrorString(result, &errorStr);
            fprintf(stderr, "Failed to get function %s: %s\n", kernelFunctions[i].funcName, errorStr);
            exit(EXIT_FAILURE);
        }
        printf("Successfully loaded function: %s\n", kernelFunctions[i].funcName);
    }
}





