
#include <hip/hip_runtime.h>
// kernels.cu

extern "C" __global__ void matrixAdd(int* A, int* B, int* C, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) {
        int idx = i * N + j;
        C[idx] = A[idx] + B[idx];
    }
}

extern "C" __global__ void matrixSub(int* A, int* B, int* C, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) {
        int idx = i * N + j;
        C[idx] = A[idx] - B[idx];
    }
}

extern "C" __global__ void matrixMul(int* A, int* B, int* C, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = sum;
    }
}

