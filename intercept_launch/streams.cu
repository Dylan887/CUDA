#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// 封装的函数：创建多个 CUDA 流
hipStream_t* createStreams(int numStreams) {
    // 动态分配 CUstream 数组
    hipStream_t *streams = (hipStream_t*) malloc(numStreams * sizeof(hipStream_t));
    if (streams == NULL) {
        fprintf(stderr, "Failed to allocate memory for streams.\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < numStreams; i++) {
        hipError_t result = hipStreamCreateWithFlags(&streams[i], hipStreamDefault);
        if (result != hipSuccess) {
            const char *errorStr;
            hipDrvGetErrorString(result, &errorStr);
            fprintf(stderr, "Failed to create stream %d: %s\n", i + 1, errorStr);
            exit(EXIT_FAILURE);
        } else {
            printf("Successfully created stream %d\n", i + 1);
        }
    }

    return streams;
}

// 封装销毁 CUDA 流的函数
void destroyStreams(hipStream_t *streams, int numStreams) {
    for (int i = 0; i < numStreams; i++) {
        hipError_t err = hipStreamDestroy(streams[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to destroy stream #%d: %s\n", i + 1, hipGetErrorString(err));
        } else {
            printf("Successfully destroyed stream #%d\n", i + 1);
        }
    }
}